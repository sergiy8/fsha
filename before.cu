#include "hip/hip_runtime.h"
#include "permut.h"
#include "malloc_file.c"


DATATYPE unsigned char * known[9];

PROCTYPE int Ask(TPACK pos){
    int arank = _popc(pos.b);
//  if(arank==0 || arank>8) error("Error ask");
    uint32_t  idx  = blist_get(pos.b);
    return twobit_get(known[arank] + (uint64_t)((pos.w<<arank) | pos.d) * cnk(32,arank)/4, idx);
}

PROCTYPE int StaticWhite(uint32_t w, uint32_t b, uint32_t d){
        switch(Ask(TPack((T12){w,b,d}))) {  // RANK of ask is always less the our
		case 3:
        case 0 : return R_DRA;
        case 1 : return R_WIN;
        case 2 : return R_LOS;
	default:
			return 0;
        }
}
PROCTYPE inline int MoveBlack(T12 pos){
        return StaticWhite(_brev(pos.w),_brev(pos.b),_brev(pos.d));
}


#include "move5.c"

DATATYPE unsigned char * array;
DATATYPE unsigned long long  changed[CACHESIZE];

KERNEL
#if NODAMKA
    unsigned i = ij;
    const unsigned j = 0;
#else
    unsigned i = ij >> RANK;
    unsigned j = ij & RMASK;
#endif
    unsigned busy;
    unsigned idx;
    for(idx=0,busy=ALLONE(RANK);_popc(busy)==RANK;idx++,busy = _permut(busy))
	if(twobit_get(job,idx)==0) {
   			int r;
			r = MoveWhite(TUnpack((TPACK){busy,i,j}));
			switch(r) {
			case R_NOM:
        			continue;  // rmain unknown
			case R_WIN:
					r=1;
					break;
			case R_LOS:
					r=2;
					break;
			case R_DRA:
        			r=3; // can take - but for draw position
					break;
			default:
					error("Smth wrong, r=%d",r);
			}
		twobit_set(job,idx,r);
		atomicAdd(changed+ij%CACHESIZE,1);
	}
}

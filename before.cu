#include "hip/hip_runtime.h"
#include "pack.h"
#include "permut.h"
#include "ask.c"

#ifdef DEBUG
static uint32_t fbusy,fij;
static int option_v;
#endif

PROCTYPE int StaticWhite(uint32_t w, uint32_t b, uint32_t d){
        uint32_t busy,iwhite,idamka;
        Pack(&busy,&iwhite,&idamka,w,b,d);
        switch(Ask(busy,iwhite,idamka)) {  // RANK of ask is always less the our
		case 3:
        case 0 : return 0;
        case 1 : return 5;
        case 2 : return -5;
	default:
#ifdef DEBUG
            error("%08X %X %X -> %08X %X %X",fbusy,fij>>RANK,fij&RMASK,busy,iwhite,idamka);
#endif
	return 0;
        }
}
PROCTYPE inline int MoveBlack(uint32_t w, uint32_t b, uint32_t d){
        return StaticWhite(_brev(w),_brev(b),_brev(d));
}

#include "move4.c"


DATATYPE unsigned char * array;
DATATYPE uint32_t * busylist;
DATATYPE unsigned long long  changed[CACHESIZE];

KERNEL
	unsigned ij;
	unsigned char * job  = array + abytes(RANK,idx);
	uint32_t busy = busylist[idx];

#ifdef REVERSE
	for(ij=(1<<(2*RANK))-1; ij; ij--){
#else
	for(ij=0;ij< (1<<(2*RANK)); ij++){
#endif
	if(twobit_get(job,ij)==0) {
	        uint32_t w,b,d;
   		int r;
        	Unpack(busy,ij>>RANK,ij&RMASK,&w,&b,&d);
#ifdef DEBUG
        	fbusy=busy, fij = ij;
#endif
       		r = MoveWhite(w,b,d);
        	if(r==R_NOMOVE) continue;  // rmain unknown
        	if(r<0) r=2;
        	else if(r>0) r=1;
        	else r=3; // can take - but for draw position
#ifdef DEBUG
		if( (r!=3)&&option_v)
			printf("%08X %08X %08X %d\n",w,b,d,r);
#endif
		twobit_set(job,ij,r);
		atomicAdd(changed+idx%CACHESIZE,1);
	}
	}
#if RANK >=7 // abytes(RANK,1) >= PAGE_SIZE
	munmap(job,abytes(RANK,1));
#endif
}

static void PutStat(void){
        FILE * f = fopen(STATFILE(RANK),"w");
        if(f==NULL) return;
        fprintf(f,"before\n");
        fclose(f);
}


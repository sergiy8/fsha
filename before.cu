#include "hip/hip_runtime.h"
#include "permut.h"
#include "malloc_file.c"

DATATYPE unsigned char * known[9];

PROCTYPE int Ask(TPACK pos){
    int arank = _popc(pos.b);
//  if(arank==0 || arank>8) error("Error ask");
    uint32_t  idx  = blist_get(pos.b);
    return twobit_get(known[arank] + (uint64_t)((pos.w<<arank) | pos.d) * cnk(32,arank)/4, idx);
}

PROCTYPE int StaticWhite(uint32_t w, uint32_t b, uint32_t d){
        switch(Ask(TPack((T12){w,b,d}))) {  // RANK of ask is always less the our
		case 3:
        case 0 : return 0;
        case 1 : return 5;
        case 2 : return -5;
	default:
			return 0;
        }
}
PROCTYPE inline int MoveBlack(T12 pos){
        return StaticWhite(_brev(pos.w),_brev(pos.b),_brev(pos.d));
}

#include "move4.c"


DATATYPE unsigned char * array;
DATATYPE unsigned long long  changed[CACHESIZE];

KERNEL
#if NODAMKA
    unsigned i = ij;
    const unsigned j = 0;
#else
    unsigned i = ij >> RANK;
    unsigned j = ij & RMASK;
#endif
    unsigned busy;
    unsigned idx;
    for(idx=0,busy=ALLONE(RANK);_popc(busy)==RANK;idx++,busy = _permut(busy))
	if(twobit_get(job,idx)==0) {
   			int r;
			r = MoveWhite(TUnpack((TPACK){busy,i,j}));
        	if(r==R_NOMOVE) continue;  // rmain unknown
        	if(r<0) r=2;
        	else if(r>0) r=1;
        	else r=3; // can take - but for draw position
		twobit_set(job,idx,r);
		atomicAdd(changed+ij%CACHESIZE,1);
	}
}

static void PutStat(void){
        FILE * f = fopen(STATFILE(RANK),"w");
        if(f==NULL) return;
        fprintf(f,"before\n");
        fclose(f);
}


#include "hip/hip_runtime.h"
#include "pack.h"
#include "permut.h"
#include "ask.c"

PROCTYPE int StaticWhite(uint32_t w, uint32_t b, uint32_t d){
        uint32_t busy,iwhite,idamka;
        Pack(&busy,&iwhite,&idamka,w,b,d);
        switch(Ask(busy,iwhite,idamka)) {  // RANK of ask is always less the our
		case 3:
        case 0 : return 0;
        case 1 : return 5;
        case 2 : return -5;
	default:
			return 0;
        }
}
PROCTYPE inline int MoveBlack(uint32_t w, uint32_t b, uint32_t d){
        return StaticWhite(_brev(w),_brev(b),_brev(d));
}

#include "move4.c"


DATATYPE unsigned char * array;
DATATYPE unsigned long long  changed[CACHESIZE];

KERNEL
    unsigned i = ij >> RANK;
    unsigned j = ij & RMASK;
    unsigned busy;
    unsigned idx;
    unsigned char * job = array + ij * CNK /4;
    for(idx=0,busy=ALLONE(RANK);_popc(busy)==RANK;idx++,busy = _permut(busy))
	if(twobit_get(job,idx)==0) {
	        uint32_t w,b,d;
   			int r;
        	Unpack(busy,i,j,&w,&b,&d);
       		r = MoveWhite(w,b,d);
        	if(r==R_NOMOVE) continue;  // rmain unknown
        	if(r<0) r=2;
        	else if(r>0) r=1;
        	else r=3; // can take - but for draw position
		twobit_set(job,idx,r);
		atomicAdd(changed+ij%CACHESIZE,1);
	}
}

static void PutStat(void){
        FILE * f = fopen(STATFILE(RANK),"w");
        if(f==NULL) return;
        fprintf(f,"before\n");
        fclose(f);
}


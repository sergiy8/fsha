#include "hip/hip_runtime.h"
#include "facecontrol.h"

DATATYPE unsigned char * array;
DATATYPE unsigned long long  changed[CACHESIZE];

KERNEL
#if NODAMKA
	unsigned i = ij;
	const unsigned j = 0;
#else
	unsigned i = ij >> RANK;
	unsigned j = ij & RMASK;
#endif
	unsigned busy;
	unsigned idx;
	for(idx=0,busy=ALLONE(RANK);_popc(busy)==RANK;idx++,busy = _permut(busy)){
		if(FaceControl(busy,i,j)) {
		   twobit_set(job,idx,3);
		   atomicAdd(changed + ij%CACHESIZE,1);
			continue;
		}
		if ( i == 0) {
		   twobit_set(job,idx,2);
			continue;
		}
		if ( i == RMASK) {
		   twobit_set(job,idx,1);
			continue;
		}
	}
}

static void PutStat(void){
        FILE * f = fopen(STATFILE(RANK),"w");
        if(f==NULL) return;
        fprintf(f,"mk_data\n");
        fclose(f);
}


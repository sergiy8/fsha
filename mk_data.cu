#include "hip/hip_runtime.h"
#include "facecontrol.h"

DATATYPE unsigned char * array;
DATATYPE uint32_t * busylist;
DATATYPE unsigned long long  changed[CACHESIZE];

KERNEL
	int i;
	unsigned char * job  = array + abytes(RANK,idx);
	uint32_t busy = busylist[idx];

	for(i=0;i<(1<<RANK);i++) {
		twobit_set(job,i,2); // no white - lose
		twobit_set(job,(RMASK<<RANK)+i,1);
	}
	for(i=0;i<(1<<(2*RANK));i++)
		if(FaceControl(busy,i>>RANK,i&RMASK)){
		   twobit_set(job,i,3);
		   atomicAdd(changed + idx%CACHESIZE,1);
		}
}

static void PutStat(void){
        FILE * f = fopen(STATFILE(RANK),"w");
        if(f==NULL) return;
        fprintf(f,"mk_data\n");
        fclose(f);
}


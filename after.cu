#include "hip/hip_runtime.h"
#include "facecontrol.h"

DATATYPE unsigned char * array;
DATATYPE uintmax_t changed[CACHESIZE];
KERNEL
    unsigned i = ij >> RANK;
    unsigned j = ij & RMASK;
    unsigned busy;
    unsigned idx;
    unsigned char * job = array + ij * CNK /4;
    for(idx=0,busy=ALLONE(RANK);_popc(busy)==RANK;idx++,busy = _permut(busy)){
		if(FaceControl(busy,i,j)==0)
		if(twobit_get(job,idx)==3){
			twobit_clear(job,idx);
			atomicAdd(changed+ij%CACHESIZE,1);
		}
	}
}

#include <unistd.h>
static void PutStat(void){
	unlink(STATFILE(RANK));
}


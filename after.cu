#include "hip/hip_runtime.h"
#include "facecontrol.h"

DATATYPE unsigned char * array;
DATATYPE unsigned * busylist;
DATATYPE uintmax_t changed[CACHESIZE];

KERNEL
	unsigned ij;
	unsigned char * job  = array + abytes(RANK,idx);
	uint32_t busy = busylist[idx];

	for(ij=0;ij<(1<<2*RANK);ij++){
		if(FaceControl(busy,ij>>RANK,ij&RMASK)==0)
		if(twobit_get(job,ij)==3){
			twobit_clear(job,ij);
			atomicAdd(changed+idx%CACHESIZE,1);
		}
	}
}

#include <unistd.h>
static void PutStat(void){
	unlink(STATFILE(RANK));
}


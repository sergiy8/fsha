#include "hip/hip_runtime.h"
#include "pack.h"

DATATYPE unsigned char * array;
DATATYPE unsigned long long  changed[CACHESIZE];


PROCTYPE int StaticWhite(uint32_t w, uint32_t b, uint32_t d){
        uint32_t busy,iwhite,idamka;
	uint32_t idx;
        Pack(&busy,&iwhite,&idamka,w,b,d);
		idx = blist_get(busy);
        switch(twobit_get(array + (uint64_t)((iwhite<<RANK)|idamka) * JOB_SIZE, idx)){
        case 3 : // Cimus ZZ
		return 0;
        case 0 :
		return 0;
        case 1 :
		return 5;
        case 2 :
		return -5;
        default:
		return 0;
        }
}
PROCTYPE inline int MoveBlack(uint32_t w, uint32_t b, uint32_t d){
        return StaticWhite(_brev(w),_brev(b),_brev(d));
}

static unsigned spewcount[CACHESIZE];
#define SPEW_LEVEL 128

#include "move4.c"
KERNEL
    unsigned i = ij >> RANK;
    unsigned j = ij & RMASK;
    unsigned busy;
    unsigned idx;
    for(idx=0,busy=ALLONE(RANK);_popc(busy)==RANK;idx++,busy = _permut(busy))
	if(twobit_get(job,idx)==0) {
	        uint32_t w,b,d;
   		int r;
        	Unpack(busy,i,j,&w,&b,&d);
       		r = MoveWhite(w,b,d);
		if(r==0) continue;
		twobit_set(job,idx,r<0?2:1);
		if ( spewcount[ij%CACHESIZE]++ % SPEW_LEVEL == 0 )
			fprintf(stderr,"%08X %X %x %d\n",busy,i,j,r<0?2:1);
		atomicAdd(changed+ij%CACHESIZE,1);
	}
}

static void PutStat(void){
	FILE * f = fopen(STATFILE(RANK),"w");
	if(f==NULL) return;
#ifdef WRANK
	fprintf(f,"klini WRANK=%d\n",WRANK);
#else
	fprintf(f,"klini\n");
#endif
	fclose(f);
}

#include "hip/hip_runtime.h"
#include "pack.h"

#include "search.c"

DATATYPE unsigned char * array;
DATATYPE uint32_t * busylist;
DATATYPE unsigned long long  changed[CACHESIZE];

#ifdef DEBUG
static uint32_t f1,f2,f3;
static int option_v;

//#define F1 0x88000000
#define F1 0x80000001
#define F2 1
#define F3 3
#define pprintf(args...) do{if((f1==F1)&&(f2==F2)&&(f3==F3)) {printf("%d:",__LINE__);printf(args);};}while(0)
#else
#define pprintf(args...) do{}while(0)
#endif


PROCTYPE int StaticWhite(uint32_t w, uint32_t b, uint32_t d){
        uint32_t busy,iwhite,idamka;
	uint32_t idx;
        Pack(&busy,&iwhite,&idamka,w,b,d);
	idx = search(busy,busylist,CNK);
        switch(twobit_get(array + abytes(RANK,idx),(iwhite<<RANK)|idamka)){
        case 3 : // Cimus ZZ
		return 0;
        case 0 :
		return 0;
        case 1 :
		return 5;
        case 2 :
		return -5;
        default:
#ifdef DEBUG
		error("%08X %X %X -> %08X %08X %08X",f1,f2,f3,busy,iwhite,idamka);
#endif
		return 0;
        }
}
PROCTYPE inline int MoveBlack(uint32_t w, uint32_t b, uint32_t d){
        return StaticWhite(_brev(w),_brev(b),_brev(d));
}

#include "move4.c"

KERNEL
	unsigned ij;
	unsigned char * job  = array + abytes(RANK,idx);
	uint32_t busy = busylist[idx];

	for(ij=0; ij< (1<<2*RANK); ij++) {
#ifdef WRANK
	int wrank = _popc(ij>>RANK);
	if( (wrank!=WRANK) && (wrank!= (RANK-WRANK))) continue;
#endif
	if(twobit_get(job,ij)==0) {
	        uint32_t w,b,d;
   		int r;
#ifdef DEBUG
        	f1=busy,f2=ij>>RANK,f3=ij&RMASK;
#endif
        	Unpack(busy,ij>>RANK,ij&RMASK,&w,&b,&d);
       		r = MoveWhite(w,b,d);
		if(r==0) continue;
#ifdef DEBUG
		if(option_v)
			printf("%08X %X %X %d\n",busy,i,j,r);
#endif
		twobit_set(job,ij,r<0?2:1);
		atomicAdd(changed+idx%CACHESIZE,1);
	}
	}
}

static void PutStat(void){
	FILE * f = fopen(STATFILE(RANK),"w");
	if(f==NULL) return;
#ifdef WRANK
	fprintf(f,"klini WRANK=%d\n",WRANK);
#else
	fprintf(f,"klini\n");
#endif
	fclose(f);	
}

#include "hip/hip_runtime.h"

DATATYPE unsigned char * array;
DATATYPE unsigned long long  changed[CACHESIZE];


PROCTYPE int StaticWhite(uint32_t w, uint32_t b, uint32_t d){
		TPACK x = TPack((T12){w,b,d});
	uint32_t idx;
#if NODAMKA
	if(idamka){
		printf("%08X %X %X\n",x.b,x.w,x.d);
		return 0;
	}
#endif
		idx = blist_get(x.b);
#if NODAMKA
        switch(twobit_get(array + (uint64_t)iwhite * JOB_SIZE, idx)){
#else
        switch(twobit_get(array + (uint64_t)((x.w<<RANK)|x.d) * JOB_SIZE, idx)){
#endif
        case 3 : // Cimus ZZ
		return R_DRA;
        case 0 :
		return R_DRA;
        case 1 :
		return R_WIN;
        case 2 :
		return R_LOS;
        default:
			error("Smth wrong");
        }
}
PROCTYPE inline int MoveBlack(T12 pos){
        return StaticWhite(_brev(pos.w),_brev(pos.b),_brev(pos.d));
}

static unsigned spewcount[CACHESIZE];
#define SPEW_LEVEL (1<<20)

#include "move5.c"
KERNEL
#if NODAMKA
    unsigned i = ij;
    const unsigned j = 0;
#else
    unsigned i = ij >> RANK;
    unsigned j = ij & RMASK;
#endif
    unsigned busy;
    unsigned idx;
    for(idx=0,busy=ALLONE(RANK);_popc(busy)==RANK;idx++,busy = _permut(busy))
	if(twobit_get(job,idx)==0) {
   		int r;
		r = MoveWhite(TUnpack((TPACK){busy,i,j}));
		switch(r) {
		case R_DRA:
			continue;
		case R_WIN:
			twobit_set(job,idx,1);
			break;
		case R_NOM:
		case R_LOS:
			twobit_set(job,idx,2);
			break;
		default:
			error("Smth wrong, r=%d",r);
		}
		if ( spewcount[ij%CACHESIZE]++ % SPEW_LEVEL == 0 )
			fprintf(stderr,"%08X %X %x %d\n",busy,i,j,r==R_WIN?1:2);
		atomicAdd(changed+ij%CACHESIZE,1);
	}
}

#include "hip/hip_runtime.h"

DATATYPE unsigned char * array;
DATATYPE unsigned long long  changed[CACHESIZE];

#if KLINI_MEGASK
#include "megask.c"
#endif

#if NODAMKA
static FILE * pf; // Text with unknown combinations
static int option_q;
void pf_init(void){
	char fname[PATH_MAX];
	snprintf(fname,sizeof(fname),DATADIR "%d-q%d.unsorted", RANK, option_q);
	pf = fopen(fname,"w");
	if( pf == NULL)
		error ("Cannot fopen %s",fname);
}
#endif


PROCTYPE int StaticWhite(uint32_t w, uint32_t b, uint32_t d){
		TPACK x = TPack((T12){w,b,d});
#if KLINI_MEGASK
	switch(megask(x)) {
		case ASK_DRAW:
		case ASK_NODB:
		case 3:
			return 0;
		case ASK_WHITE:
			return 5;
		case ASK_BLACK:
			return -5;
		default:
			error("Smth wrong: %08X %X %X = %d", x.b, x.w, x.d, megask(x));
	}
#else
#if NODAMKA
	if(x.d)
	if(option_q){
		if( fwrite(&x,sizeof(x),1,pf) != 1)
			error("fwrite()");
		return 0;
	}
#endif
		uint32_t idx = blist_get(x.b);
#if NODAMKA
        switch(twobit_get(array + (uint64_t)x.w * JOB_SIZE, idx)){
#else
        switch(twobit_get(array + (uint64_t)((x.w<<RANK)|x.d) * JOB_SIZE, idx)){
#endif
        case 3 : // Cimus ZZ
		return R_DRA;
        case 0 :
		return R_DRA;
        case 1 :
		return R_WIN;
        case 2 :
		return R_LOS;
        default:
			error("Smth wrong");
        }
#endif // KLINI_MEGASK
}
PROCTYPE inline int MoveBlack(T12 pos){
        return StaticWhite(_brev(pos.w),_brev(pos.b),_brev(pos.d));
}

static unsigned spewcount[CACHESIZE];
#define SPEW_LEVEL (1<<20)

#include "move5.c"
KERNEL
#if NODAMKA
    unsigned i = ij;
    const unsigned j = 0;
#else
    unsigned i = ij >> RANK;
    unsigned j = ij & RMASK;
#endif
    unsigned busy;
    unsigned idx;
    for(idx=0,busy=ALLONE(RANK);_popc(busy)==RANK;idx++,busy = _permut(busy))
	if(twobit_get(job,idx)==0) {
   		int r;
		r = MoveWhite(TUnpack((TPACK){busy,i,j}));
		switch(r) {
		case R_DRA:
			continue;
		case R_WIN:
			twobit_set(job,idx,1);
			break;
		case R_NOM:
		case R_LOS:
			twobit_set(job,idx,2);
			break;
		default:
			error("Smth wrong, r=%d",r);
		}
		if ( spewcount[ij%CACHESIZE]++ % SPEW_LEVEL == 0 )
			fprintf(stderr,"%08X %X %x %d\n",busy,i,j,r==R_WIN?1:2);
		atomicAdd(changed+ij%CACHESIZE,1);
	}
}

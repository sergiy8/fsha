#include "hip/hip_runtime.h"

DATATYPE unsigned char * array;
DATATYPE unsigned long long  changed[CACHESIZE];


PROCTYPE int StaticWhite(uint32_t w, uint32_t b, uint32_t d){
		TPACK x = TPack((T12){w,b,d});
	uint32_t idx;
#if NODAMKA
	if(idamka){
		printf("%08X %X %X\n",x.b,x.w,x.d);
		return 0;
	}
#endif
		idx = blist_get(x.b);
#if NODAMKA
        switch(twobit_get(array + (uint64_t)iwhite * JOB_SIZE, idx)){
#else
        switch(twobit_get(array + (uint64_t)((x.w<<RANK)|x.d) * JOB_SIZE, idx)){
#endif
        case 3 : // Cimus ZZ
		return 0;
        case 0 :
		return 0;
        case 1 :
		return 5;
        case 2 :
		return -5;
        default:
		return 0;
        }
}
PROCTYPE inline int MoveBlack(T12 pos){
        return StaticWhite(_brev(pos.w),_brev(pos.b),_brev(pos.d));
}

static unsigned spewcount[CACHESIZE];
#define SPEW_LEVEL (1<<0)

#include "move4.c"
KERNEL
#if NODAMKA
    unsigned i = ij;
    const unsigned j = 0;
#else
    unsigned i = ij >> RANK;
    unsigned j = ij & RMASK;
#endif
    unsigned busy;
    unsigned idx;
    for(idx=0,busy=ALLONE(RANK);_popc(busy)==RANK;idx++,busy = _permut(busy))
	if(twobit_get(job,idx)==0) {
   		int r;
		r = MoveWhite(TUnpack((TPACK){busy,i,j}));
		if(r==0) continue;
		twobit_set(job,idx,r<0?2:1);
		if ( spewcount[ij%CACHESIZE]++ % SPEW_LEVEL == 0 )
			fprintf(stderr,"%08X %X %x %d\n",busy,i,j,r<0?2:1);
		atomicAdd(changed+ij%CACHESIZE,1);
	}
}

static void PutStat(void){
	FILE * f = fopen(STATFILE(RANK),"w");
	if(f==NULL) return;
#ifdef WRANK
	fprintf(f,"klini WRANK=%d\n",WRANK);
#else
	fprintf(f,"klini\n");
#endif
	fclose(f);
}

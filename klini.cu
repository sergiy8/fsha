#include "hip/hip_runtime.h"

DATATYPE unsigned char * array;
DATATYPE unsigned long long  changed[CACHESIZE];

#if NODAMKA
#undef IN_klini
#include "megask.c"
#define IN_klini 1
#include <pthread.h>
static pthread_key_t key;
static int question(TPACK x) {
	FILE * f = pthread_getspecific(key);
	if (fwrite(&x,sizeof(x),1,f) != 1)
		error("%m");
	return R_UNK;
}
typedef void (*DESTRUCTOR)(void *);
#define EXTRA_INIT     megask_init(); pthread_key_create(&key, (DESTRUCTOR)fclose);
#endif

PROCTYPE int StaticWhite(uint32_t w, uint32_t b, uint32_t d){
		TPACK x = TPack((T12){w,b,d});
#if NODAMKA
	switch(megask(x)) {
		case ASK_NODB:
			if(d)
				return question(x);
		case ASK_UNK:
			return R_UNK;
		case ASK_DRAW:
			return R_DRA;
		case ASK_WHITE:
			return R_WIN;
		case ASK_BLACK:
			return R_LOS;
		default:
			error("Smth wrong: %08X %X %X = %d", x.b, x.w, x.d, megask(x));
	}
#else
		uint32_t idx = blist_get(x.b);
        switch(twobit_get(array + (uint64_t)((x.w<<RANK)|x.d) * JOB_SIZE, idx)){
        case 3 : // Cimus ZZ
		return R_DRA;
        case 0 :
		return R_DRA;
        case 1 :
		return R_WIN;
        case 2 :
		return R_LOS;
        default:
			error("Smth wrong");
        }
#endif
}
PROCTYPE inline int MoveBlack(T12 pos){
        return StaticWhite(_brev(pos.w),_brev(pos.b),_brev(pos.d));
}

static unsigned spewcount[CACHESIZE];
#define SPEW_LEVEL (1<<20)

#include "move5.c"
KERNEL
#if NODAMKA
    unsigned i = ij;
    const unsigned j = 0;
#else
    unsigned i = ij >> RANK;
    unsigned j = ij & RMASK;
#endif
    unsigned busy;
    unsigned idx;
    for(idx=0,busy=ALLONE(RANK);_popc(busy)==RANK;idx++,busy = _permut(busy))
	if(twobit_get(job,idx)==0) {
   		int r;
		r = MoveWhite(TUnpack((TPACK){busy,i,j}));
		switch(r) {
		case R_UNK:
		case R_DRA:
			continue;
		case R_WIN:
			twobit_set(job,idx,1);
			break;
		case R_NOM:
		case R_LOS:
			twobit_set(job,idx,2);
			break;
		default:
			error("Smth wrong, r=%d",r);
		}
		if ( spewcount[ij%CACHESIZE]++ % SPEW_LEVEL == 0 )
			fprintf(stderr,"%08X %X %x %d\n",busy,i,j,r==R_WIN?1:2);
		atomicAdd(changed+ij%CACHESIZE,1);
	}
}
